#include "hip/hip_runtime.h"
#include "system.cuh"
#include "Rand.hpp"
#include <vector>

double g_beta;
double t_param;
double U_param;
double mu_param;

void init_state(){
    suqa::init_state();
//	suqa::apply_x(bm_spin[1]);
//	suqa::apply_h(bm_spin[1]);
//	suqa::apply_cx(bm_spin[1], bm_spin[0]);
}

void init_state_testevo(){
    size_t dim = 1 << syst_qbits;

    std::vector<double> re_coeff(dim, 0.0);
    std::vector<double> im_coeff(dim, 0.0);

    double re_coeff_1 = 0.435162146493599;
    double re_coeff_2 = 0.557345410189304;

    re_coeff[5] = re_coeff_1;
    re_coeff[10] = re_coeff_1;
    re_coeff[6] = re_coeff_2;
    re_coeff[9] = re_coeff_2;

    suqa::init_state(re_coeff,im_coeff);
}


void exp_it_id_x_x(const bmReg& q, uint pos_id, double phase_t){

	suqa::apply_pauli_TP_rotation({q[(pos_id+1)%4],q[(pos_id+2)%4]}, {PAULI_X,PAULI_X}, phase_t);

}

// applies exp(-iHt)
void evolution(const double& t, const int& n){
    //(void)n;
    double N = (double) n;

    double fac_t = -t_param/2.0;
    double fac_U = U_param/4.0;
    double fac_mu = mu_param/2.0 - U_param/4.0;
    double fac_phase = (U_param/2.0 - 2.0*mu_param);

    fac_t = fac_t/N;
    fac_U = fac_U/N;
    fac_mu = fac_mu/N;
    fac_phase = fac_phase/N;

for (int it=0; it<N; it++) {
    // alpha term
    suqa::apply_pauli_TP_rotation({bm_spin[0],bm_spin[1]}, {PAULI_X,PAULI_X}, -t*fac_t);
    suqa::apply_pauli_TP_rotation({bm_spin[0],bm_spin[1]}, {PAULI_Y,PAULI_Y}, -t*fac_t);
    suqa::apply_pauli_TP_rotation({bm_spin[2],bm_spin[3]}, {PAULI_X,PAULI_X}, -t*fac_t);
    suqa::apply_pauli_TP_rotation({bm_spin[2],bm_spin[3]}, {PAULI_Y,PAULI_Y}, -t*fac_t);

    // beta term
    suqa::apply_pauli_TP_rotation({bm_spin[0],bm_spin[2]}, {PAULI_Z,PAULI_Z}, -t*fac_U);
    suqa::apply_pauli_TP_rotation({bm_spin[1],bm_spin[3]}, {PAULI_Z,PAULI_Z}, -t*fac_U);

    // gamma term
    suqa::apply_pauli_TP_rotation({bm_spin[0]}, {PAULI_Z}, -t*fac_mu);
    suqa::apply_pauli_TP_rotation({bm_spin[1]}, {PAULI_Z}, -t*fac_mu);
    suqa::apply_pauli_TP_rotation({bm_spin[2]}, {PAULI_Z}, -t*fac_mu);
    suqa::apply_pauli_TP_rotation({bm_spin[3]}, {PAULI_Z}, -t*fac_mu);

    // global phase
//    for (int s=0; s<4; s++) {
//    suqa::apply_x(bm_spin[s]);
//    suqa::apply_u1(bm_spin[s], -t*fac_phase);
//    suqa::apply_x(bm_spin[s]);
//    suqa::apply_u1(bm_spin[s], -t*fac_phase);
//    }

}

//	for (uint iii=0; iii<4; ++iii){
//		exp_it_id_x_x(bm_spin, iii, -t);
// 	}
}

// qsa specifics
void qsa_init_state(){
    suqa::init_state();
    suqa::apply_h(bm_spin[0]);
    suqa::apply_h(bm_spin[1]);
    suqa::apply_h(bm_spin[2]);
    suqa::apply_cx(bm_spin[0], bm_spin_tilde[0]);
    suqa::apply_cx(bm_spin[1], bm_spin_tilde[1]);
    suqa::apply_cx(bm_spin[2], bm_spin_tilde[2]);
}

void evolution_szegedy(const double& t, const int& n){
    (void)n;
      DEBUG_CALL(std::cout<<"before evolution_szegedy()"<<std::endl);
      DEBUG_READ_STATE();
      DEBUG_CALL(std::cout<<"apply evolution_szegedy()"<<std::endl);
    for (uint i = 0; i < 3; i++) {
      suqa::apply_pauli_TP_rotation({bm_spin_tilde[(0+i)%3],bm_spin_tilde[(1+i)%3]}, {PAULI_X,PAULI_X}, -t);
      DEBUG_CALL(std::cout<<"apply pauli 1 it "<<i<<std::endl);
      DEBUG_READ_STATE();
      suqa::apply_pauli_TP_rotation({bm_spin[(0+i)%3],bm_spin[(1+i)%3]}, {PAULI_X,PAULI_X}, t);
      DEBUG_CALL(std::cout<<"apply pauli 2 it "<<i<<std::endl);
      DEBUG_READ_STATE();
    }
}

void evolution_measure(const double& t, const int& n){
    (void)n;
  for (uint i = 0; i < 3; i++) {
    suqa::apply_pauli_TP_rotation({bm_spin[(0+i)%3],bm_spin[(1+i)%3]}, {PAULI_X,PAULI_X}, -t);
  }

}
void evolution_tracing(const double& t, const int& n){
    (void)n;
  for (uint i = 0; i < 3; i++) {
    suqa::apply_pauli_TP_rotation({bm_spin_tilde[(0+i)%3],bm_spin_tilde[(1+i)%3]}, {PAULI_X,PAULI_X}, -t);
  }

}

/* Measure facilities */
const uint op_bits = 4; // 2^op_bits is the number of eigenvalues for the observable
const bmReg bm_op = bm_spin; // where the measure has to be taken
const std::vector<double> op_vals = {0.0, 1.0, 1.0, 2.0, 1.0, 2.0, 2.0, 3.0, 1.0, 2.0, 2.0, 3.0, 2.0, 3.0, 3.0, 4.0}; // eigvals
 
// change basis to the observable basis somewhere in the system registers
void apply_measure_rotation(){
//	suqa::apply_h(bm_spin[0]);
//	suqa::apply_h(bm_spin[1]);
//	suqa::apply_h(bm_spin[2]);
//	suqa::apply_cx(bm_spin[0], bm_spin[1]);
//	suqa::apply_cx(bm_spin[0], bm_spin[2]);
//	suqa::apply_u1(bm_spin[2], M_PI*0.5);
}

// inverse of the above function
void apply_measure_antirotation(){
 	apply_measure_rotation();
}

// map the classical measure recorded in creg_vals
// to the corresponding value of the observable;
// there is no need to change it
double get_meas_opvals(const uint& creg_vals){
    return op_vals[creg_vals];
}

// actually perform the measure
// there is no need to change it
double measure_X(pcg& rgen){

    std::vector<uint> classics(op_bits);
    
    apply_measure_rotation();

    std::vector<double> rdoubs(op_bits);
    for(auto& el : rdoubs){
        el = rgen.doub();
    }
    suqa::measure_qbits(bm_op, classics, rdoubs);

    apply_measure_antirotation();

    uint meas = 0U;
    for(uint i=0; i<op_bits; ++i){
        meas |= (classics[i] << i);
    }

    return get_meas_opvals(meas);

    return 0.0;
}

/* Moves facilities */

std::vector<double> C_weigthsums = {1./4, 2./4, 3./4, 1.0};

void apply_C(const uint &Ci){
    if(Ci>3)
        throw std::runtime_error("ERROR: wrong move selection");
    //suqa::apply_x(bm_spin[Ci]);
    suqa::apply_h(bm_spin[Ci]);
}

void apply_C_inverse(const uint &Ci){
    apply_C(Ci);
}

void qsa_apply_C(const uint &Ci){
  if(Ci>2) throw std::runtime_error("ERROR: wrong move selection");
  suqa::apply_h(bm_spin_tilde[Ci]);
// suqa::apply_h(state,bm_spin_tilde[(Ci+1)%3]);


  // suqa::apply_h(state,bm_spin_tilde);
}

void qsa_apply_C_inverse(const uint &Ci){
  if(Ci>2) throw std::runtime_error("ERROR: wrong move selection");
  //suqa::apply_h(state,bm_spin_tilde);
  //suqa::apply_h(state,bm_spin_tilde[(Ci+1)%3]);
  suqa::apply_h(bm_spin_tilde[Ci]);
}

std::vector<double> get_C_weigthsums(){ return C_weigthsums; }

